/**
 * Nathan Dunn
 * Project 3: Work Efficient Parallel Reduction and Work Efficient Parallel Prefix Sum
 * Professor Liu
 * CS-4370-90
 * 11-18-19
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define  N 16// Length of vector that will be summed
#define BLOCK_SIZE 4// size of thread blocks

/**
 * Performs CPU Sum Reduction
 * x: vector to be summed
 * length: width of vector x
*/
int hostSumReduction(int* x, int length){

	for (int i = 1; i < length; i++)
		x[0] =x [0] + x[i];

	int overallSum = x[0];

	return overallSum;
}

/**
 * Performs GPU Sum Reduction
 * a: vector to be summed
 * length: Length of the vector to be added
*/
__global__ void deviceSumReduction(int *input, int *sum, int length){
	__shared__ int partialSum[2*BLOCK_SIZE];
	unsigned int tx = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;

	partialSum[tx] = input[start + tx];
	partialSum[blockDim.x+tx] = input[start+ blockDim.x+tx];

	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2){
		__syncthreads();
		if (tx < stride)
			partialSum[tx] += partialSum[tx+stride];
	}
	
	sum[blockIdx.x] = partialSum[0];
	printf("(%d, %d)\t%d\n", blockIdx.x, threadIdx.x, sum[blockIdx.x]); 
}



int main(void){
	int *a, *b, *dev_a, *dev_b;
	
	// used to keep track of sum array size (if n is larger than 2 * blocksize), then 
	// multiple kernel calls will have to be made
	int sumSize = N / (2 * BLOCK_SIZE); 
	
	// block and grid initialization for gpu
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid(ceil(N / dimBlock.x), 1, 1);
	
	// allocate vectors for cpu
	a = (int*)malloc(sizeof(int)* N);
	b = (int*)malloc(sizeof(int)* sumSize);
	
	// allocate vectors for gpu
	hipMalloc((void **)(&dev_a), N* sizeof(int));
	hipMalloc((void **)(&dev_b), sumSize * sizeof(int));
	
	// initialize vector
	int init =1325;
	for(int i=0;i<N;i++){
		// init=3125*init%65521;
		// a[i]=(init-32768)/16384;
		a[i] = i;
	}
	
	// copy array a (host) to dev_a (device)
	hipMemcpy(dev_a,a,N * sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	// Launch kernels for initial reduction
	deviceSumReduction<<<dimGrid, dimBlock>>>(dev_a, dev_b, N);
	hipDeviceSynchronize();
	
	// copy results from gpu back to host
	hipMemcpy(b, dev_b, sumSize * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	// run sum reducton on host
	int sum = hostSumReduction(a, N);
	
	printf("Hello world!\n");
	printf("%d\n", sum);
	
	//debugging b
	for(int i =0; i < sumSize; i++){
		printf("%d ", b[i]);
	}
	printf("\n");
	
	
	// free system and device memory
	free(a);
	free(b);
	hipFree(dev_a);
	hipFree(dev_b);
	
	
	return 0;
}