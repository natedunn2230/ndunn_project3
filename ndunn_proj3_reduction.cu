#include "hip/hip_runtime.h"
/**
 * Nathan Dunn
 * Project 3: Work Efficient Parallel Reduction and Work Efficient Parallel Prefix Sum
 * Professor Liu
 * CS-4370-90
 * 11-18-19
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define  N 16777216  // Length of vector that will be summed
#define BLOCK_SIZE 128// size of thread blocks

/**
 * Performs CPU Sum Reduction
 * x: vector to be summed
 * length: width of vector x
*/
int hostSumReduction(int* x, int length){
	for (int i = 1; i < length; i++)
		x[0] =x [0] + x[i];

	int overallSum = x[0];

	return overallSum;
}

/**
 * Performs GPU Sum Reduction
 * input: vector to be summed
 * length: Length of the vector to be added
*/
__global__ void deviceSumReduction(int *input, int length){
	__shared__ int partialSum[2*BLOCK_SIZE];
	unsigned int tx = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;

	partialSum[tx] = input[start + tx];
	partialSum[blockDim.x+tx] = input[start+ blockDim.x+tx];

	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2){
		__syncthreads();
		if (tx < stride)
			partialSum[tx] += partialSum[tx+stride];
	}
	
	input[blockIdx.x] = partialSum[0];
	//printf("(%d, %d)\t%d\n", blockIdx.x, threadIdx.x, input[blockIdx.x]); 
}

/**
 * Calls cuda kernel function recursively to get total sum reduction
 * a: Array to be summed
 * length: length of array to be summed
*/
void applyReduction(int *vect, int length, float *gpuTimes){
	hipEvent_t gpuStart,gpuStop;
	
	// holds each time for computation / copy of each kernel call
	float copyTo, computationTime, copyFrom;
	
	int *vect_dev;
	int sumSize = ceil((float)length / (2 * BLOCK_SIZE)); // size of sum array after each iteration
	
	// block and grid initialization for gpu
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid(ceil((float)length / dimBlock.x), 1, 1);
	
	// allocate vectors for gpu
	hipMalloc((void **)(&vect_dev), length * sizeof(int));
	
	// Begin measuring time for copying memory over to device
	// hipEventCreate(&gpuStart);
	// hipEventCreate(&gpuStop);
	// hipEventRecord(gpuStart,0);
	
	// copy array a (host) to dev_a (device)
	hipMemcpy(vect_dev, vect, length * sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	// Finish measuring time for copying memory over to device
	// hipEventRecord(gpuStop,0);
	// hipEventSynchronize(gpuStop);
	// hipEventElapsedTime(&copyTo,gpuStart,gpuStop);
    // hipEventDestroy(gpuStart);
    // hipEventDestroy(gpuStop);
	
	
	// Begin measuring GPU computation time
	// hipEventCreate(&gpuStart);
	// hipEventCreate(&gpuStop);
	// hipEventRecord(gpuStart,0);
	
	// Launch kernels for reduction
	deviceSumReduction<<<dimGrid, dimBlock>>>(vect_dev, length);
	hipDeviceSynchronize();
	
	// Finish measuring GPU computation time
	// hipEventRecord(gpuStop,0);
	// hipEventSynchronize(gpuStop);
	// hipEventElapsedTime(&computationTime,gpuStart,gpuStop);
    // hipEventDestroy(gpuStart);
    // hipEventDestroy(gpuStop);
	
	// Begin measuring time for copying memory back to host
	// hipEventCreate(&gpuStart);
	// hipEventCreate(&gpuStop);
	// hipEventRecord(gpuStart,0);
	
	// copy results from gpu back to host
	hipMemcpy(vect, vect_dev, length * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	// Finish measuring time for copying memory back to host
	// hipEventRecord(gpuStop,0);
	// hipEventSynchronize(gpuStop);
	// hipEventElapsedTime(&copyFrom,gpuStart,gpuStop);
    // hipEventDestroy(gpuStart);
    // hipEventDestroy(gpuStop);
	
	// free allocated device memory
	hipFree(vect_dev);
	hipDeviceSynchronize();
	
	printf("\nFOR sum size of %d\n", sumSize);
	for(int i = 0; i < sumSize; i++){
		
		printf("%d ", vect[i]);
	}
	printf("\n");
	
	// update total times
	// gpuTimes[0] += copyTo;
	// gpuTimes[1] += computationTime;
	// gpuTimes[2] += copyFrom;
	
	// apply reduction again on sum array, if applicable
	if(sumSize > 1)
		return applyReduction(vect, sumSize, gpuTimes);
}

int main(void){
	printf("VECTOR OF SIZE: %d\nBLOCK SIZE: %d\n\n", N, BLOCK_SIZE);
	
	// passed to helper function "apply reduction" to get transfer to time [0],
	// computation time [1] and transfer back time [2]
	float gpuTimes[3];

	// allocate vector for cpu
	int *a = (int*)malloc(sizeof(int)* N);
	int *b = (int*)malloc(sizeof(int)* N);
	
	// initialize vector
	int init =1325;
	for(int i=0;i<N;i++){
		init=3125*init%65521;
		a[i]=(init-32768)/16384;
	}
	
	memcpy(b, a, sizeof(int)* N);
	
	// run reduction on gpu device
	applyReduction(b, N, gpuTimes);
	int gpuSum = b[0];
	
	// variables used to measure cpu computation time
	clock_t cpuStart, cpuEnd;
	float cpuTimeTaken;
	
	// start measuring cpu computation time
	cpuStart = clock();
	
	// run sum reduction on host
	int cpuSum = hostSumReduction(a, N);
	
	// stop measuring cpu computation time
	cpuEnd = clock();
	cpuTimeTaken = ((float)cpuEnd - cpuStart)/CLOCKS_PER_SEC; // in seconds 
	
	
	printf("GPU SUM: %d\n", gpuSum);
	printf("CPU SUM: %d\n", cpuSum);
	
	printf("\nCPU Time: %f\n", cpuTimeTaken);
	printf("GPU Time: %f\n", gpuTimes[1]);
	printf("Memory Transfer Time: %f\n", gpuTimes[0] + gpuTimes[2]);
	
	if(cpuSum == gpuSum)
		printf("TEST PASSED!\n");
	else 
		printf("TEST FAILED!\n");
	
	// free system memory
	free(a);
	free(b);

	return 0;
}